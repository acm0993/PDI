#include "hip/hip_runtime.h"
#include "project.hpp"
#include <cstdint>

struct CudaImage
{
    uint8_t *data;
    uint32_t width;
    uint32_t height;
    size_t pitch;
};

__global__ void maxFilterTrivialKernel(CudaImage imgDst, const CudaImage imgSrc)
{
    for(int32_t j=0; j<imgSrc.height; j++)
    {
        for(int32_t i=0; i<imgSrc.width; i++)
        {
            uint8_t max = imgSrc.data[j*imgSrc.pitch + i];

            for(int32_t a=i-2; a<i+2; a++)
            {
                for(int32_t b=j-2; b<j+2; b++)
                {
                    uint8_t value=max;
                    if(a >= 0 && a < imgSrc.width && b >= 0 && b < imgSrc.height)
                        value = imgSrc.data[b*imgSrc.pitch + a];
                    if(value > max)
                        max = value;
                }
            }
            imgDst.data[j*imgDst.pitch + i] = max;
        }
    }
}

void maxFilterTrivial(lti::channel8 &res, const lti::channel8 &imgCpu, float &dt_ms)
{
    CudaImage imgSrc, imgDst;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    imgSrc.width = imgDst.width = imgCpu.columns();
    imgSrc.height = imgDst.height = imgCpu.rows();

    hipMallocPitch(&imgSrc.data, &imgSrc.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&imgDst.data, &imgDst.pitch, imgCpu.columns(), imgCpu.rows());



    hipMemcpy2D(imgSrc.data, imgSrc.pitch, imgCpu.data(), imgCpu.columns(), imgCpu.columns(), imgCpu.rows(), hipMemcpyHostToDevice);

    hipEventRecord(event1, 0);
    maxFilterTrivialKernel<<<1,1>>>(imgDst,imgSrc);
    hipEventRecord(event2, 0);


    hipEventSynchronize(event1);
    hipEventSynchronize(event2);

    hipEventElapsedTime(&dt_ms, event1, event2);

    hipMemcpy2D(res.data(), res.columns(), imgDst.data, imgDst.pitch, res.columns(), res.rows(), hipMemcpyDeviceToHost);
    hipFree(&imgSrc);
    hipFree(&imgDst);
}

__global__ void maxFilterSeparableKernel(CudaImage imgDst, const CudaImage imgSrc, CudaImage tmp)
{
  for (int32_t j = 0; j < imgSrc.height; j++) {
    for (size_t i = 0; i < imgSrc.width; i++) {
      uint8_t max = imgSrc.data[j*imgSrc.pitch + i];
      for (int32_t b = j-2; b < j+2; b++) {
        uint8_t value = max;
        if(b >= 0 && b < imgSrc.height)
          value = imgSrc.data[b*imgSrc.pitch + i];
        if(value > max)
          max = value;
      }
      tmp.data[j*tmp.pitch + i] = max;

    }

  }

  for (int32_t j = 0; j < tmp.height; j++) {
    for (size_t i = 0; i < tmp.width; i++) {
      uint8_t max = tmp.data[j*tmp.pitch + i];
      for (int32_t a = i-2; a < i+2; a++) {
        uint8_t value = max;
        if(a >= 0 && a < tmp.width)
          value = tmp.data[j*tmp.pitch + a];
        if(value > max)
          max = value;
      }
      imgDst.data[j*imgDst.pitch + i] = max;

    }
  }
}

void maxFilterSeparable(lti::channel8 &res, const lti::channel8 &imgCpu, float &dt_ms)
{
    CudaImage imgSrc, imgDst, tmp;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    imgSrc.width = imgDst.width =  tmp.width = imgCpu.columns();
    imgSrc.height = imgDst.height = tmp.height = imgCpu.rows();

    hipMallocPitch(&imgSrc.data, &imgSrc.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&imgDst.data, &imgDst.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&tmp.data, &tmp.pitch, imgCpu.columns(), imgCpu.rows());


    hipMemcpy2D(imgSrc.data, imgSrc.pitch, imgCpu.data(), imgCpu.columns(), imgCpu.columns(), imgCpu.rows(), hipMemcpyHostToDevice);

    hipEventRecord(event1, 0);
    maxFilterSeparableKernel<<<1,1>>>(imgDst,imgSrc, tmp);
    hipEventRecord(event2, 0);


    hipEventSynchronize(event1);
    hipEventSynchronize(event2);

    hipEventElapsedTime(&dt_ms, event1, event2);

    hipMemcpy2D(res.data(), res.columns(), imgDst.data, imgDst.pitch, res.columns(), res.rows(), hipMemcpyDeviceToHost);

    hipFree(&imgSrc);
    hipFree(&imgDst);
    hipFree(&tmp);
}
