#include "hip/hip_runtime.h"
#include "project.hpp"
#include <cstdint>

struct CudaImage
{
    uint8_t *data;
    uint32_t width;
    uint32_t height;
    size_t pitch;
};

#define N 16

__global__ void maxFilterTrivialKernel(CudaImage imgDst, const CudaImage imgSrc)
{
    for(int32_t j=0; j<imgSrc.height; j++)
    {
        for(int32_t i=0; i<imgSrc.width; i++)
        {
            uint8_t max = imgSrc.data[j*imgSrc.pitch + i];

            for(int32_t a=i-2; a<i+2; a++)
            {
                for(int32_t b=j-2; b<j+2; b++)
                {
                    uint8_t value=max;
                    if(a >= 0 && a < imgSrc.width && b >= 0 && b < imgSrc.height)
                        value = imgSrc.data[b*imgSrc.pitch + a];
                    if(value > max)
                        max = value;
                }
            }
            imgDst.data[j*imgDst.pitch + i] = max;
        }
    }
}

void maxFilterTrivial(lti::channel8 &res, const lti::channel8 &imgCpu, float &dt_ms)
{
    CudaImage imgSrc, imgDst;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    imgSrc.width = imgDst.width = imgCpu.columns();
    imgSrc.height = imgDst.height = imgCpu.rows();

    hipMallocPitch(&imgSrc.data, &imgSrc.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&imgDst.data, &imgDst.pitch, imgCpu.columns(), imgCpu.rows());



    hipMemcpy2D(imgSrc.data, imgSrc.pitch, imgCpu.data(), imgCpu.columns(), imgCpu.columns(), imgCpu.rows(), hipMemcpyHostToDevice);

    hipEventRecord(event1, 0);
    maxFilterTrivialKernel<<<1,1>>>(imgDst,imgSrc);
    hipEventRecord(event2, 0);


    hipEventSynchronize(event1);
    hipEventSynchronize(event2);

    hipEventElapsedTime(&dt_ms, event1, event2);

    hipMemcpy2D(res.data(), res.columns(), imgDst.data, imgDst.pitch, res.columns(), res.rows(), hipMemcpyDeviceToHost);
    hipFree(&imgSrc);
    hipFree(&imgDst);
}

__global__ void maxFilterSeparableKernel(CudaImage imgDst, const CudaImage imgSrc, CudaImage tmp)
{
  for (int32_t j = 0; j < imgSrc.height; j++) {
    for (size_t i = 0; i < imgSrc.width; i++) {
      uint8_t max = imgSrc.data[j*imgSrc.pitch + i];
      for (int32_t b = j-2; b < j+2; b++) {
        uint8_t value = max;
        if(b >= 0 && b < imgSrc.height)
          value = imgSrc.data[b*imgSrc.pitch + i];
        if(value > max)
          max = value;
      }
      tmp.data[j*tmp.pitch + i] = max;

    }

  }

  for (int32_t j = 0; j < tmp.height; j++) {
    for (size_t i = 0; i < tmp.width; i++) {
      uint8_t max = tmp.data[j*tmp.pitch + i];
      for (int32_t a = i-2; a < i+2; a++) {
        uint8_t value = max;
        if(a >= 0 && a < tmp.width)
          value = tmp.data[j*tmp.pitch + a];
        if(value > max)
          max = value;
      }
      imgDst.data[j*imgDst.pitch + i] = max;

    }
  }
}

void maxFilterSeparable(lti::channel8 &res, const lti::channel8 &imgCpu, float &dt_ms)
{
    CudaImage imgSrc, imgDst, tmp;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    imgSrc.width = imgDst.width =  tmp.width = imgCpu.columns();
    imgSrc.height = imgDst.height = tmp.height = imgCpu.rows();

    hipMallocPitch(&imgSrc.data, &imgSrc.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&imgDst.data, &imgDst.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&tmp.data, &tmp.pitch, imgCpu.columns(), imgCpu.rows());


    hipMemcpy2D(imgSrc.data, imgSrc.pitch, imgCpu.data(), imgCpu.columns(), imgCpu.columns(), imgCpu.rows(), hipMemcpyHostToDevice);


    // for (size_t i = 0; i < 10; i++) {
    //   maxFilterSeparableKernel<<<1,1>>>(imgDst,imgSrc, tmp);
    // }
    hipEventRecord(event1, 0);
    maxFilterSeparableKernel<<<1,1>>>(imgDst,imgSrc, tmp);
    hipEventRecord(event2, 0);


    hipEventSynchronize(event1);
    hipEventSynchronize(event2);

    hipEventElapsedTime(&dt_ms, event1, event2);

    hipMemcpy2D(res.data(), res.columns(), imgDst.data, imgDst.pitch, res.columns(), res.rows(), hipMemcpyDeviceToHost);

    hipFree(&imgSrc);
    hipFree(&imgDst);
    hipFree(&tmp);
}

__global__ void maxFilterSeparableMTHKernel(CudaImage imgDst, const CudaImage imgSrc, CudaImage tmp)
{
  uint8_t max;
  uint i = threadIdx.x + blockIdx.x * blockDim.x;
  uint j = threadIdx.y + blockIdx.y * blockDim.y;
  uint Idx = j*imgSrc.pitch + i;

  max = imgSrc.data[Idx];
  for (int32_t b = j-2; b < j+2; b++) {
    uint8_t value = max;
    if(b >= 0 && b < imgSrc.height)
      value = imgSrc.data[b*imgSrc.pitch + i];
    if(value > max)
      max = value;
    }
  tmp.data[Idx] = max;

  __syncthreads();
  max = tmp.data[Idx];
  for (int32_t a = i-2; a < i+2; a++) {
    uint8_t value = max;
    if(a >= 0 && a < tmp.width)
      value = tmp.data[j*tmp.pitch + a];
    if(value > max)
      max = value;
    }
  imgDst.data[Idx] = max;



}

void maxFilterSeparableMTH(lti::channel8 &res, const lti::channel8 &imgCpu, float &dt_ms)
{
    CudaImage imgSrc, imgDst, tmp;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    //uint N = 32;

    dim3 blocks(N,N);
    blocks.x = (imgCpu.columns()+N-1)/N;
    blocks.y = (imgCpu.rows()+N-1)/N;
    dim3 threads(N,N);

    imgSrc.width = imgDst.width =  tmp.width = imgCpu.columns();
    imgSrc.height = imgDst.height = tmp.height = imgCpu.rows();

    hipMallocPitch(&imgSrc.data, &imgSrc.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&imgDst.data, &imgDst.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&tmp.data, &tmp.pitch, imgCpu.columns(), imgCpu.rows());


    hipMemcpy2D(imgSrc.data, imgSrc.pitch, imgCpu.data(), imgCpu.columns(), imgCpu.columns(), imgCpu.rows(), hipMemcpyHostToDevice);

    hipEventRecord(event1, 0);
    //maxFilterSeparableMTHKernel<<<(imgCpu.columns()+N-1)/N,N>>>(imgDst,imgSrc, tmp);
    maxFilterSeparableMTHKernel<<<blocks,threads>>>(imgDst,imgSrc, tmp);
    hipEventRecord(event2, 0);


    hipEventSynchronize(event1);
    hipEventSynchronize(event2);

    hipEventElapsedTime(&dt_ms, event1, event2);

    hipMemcpy2D(res.data(), res.columns(), imgDst.data, imgDst.pitch, res.columns(), res.rows(), hipMemcpyDeviceToHost);

    hipFree(&imgSrc);
    hipFree(&imgDst);
    hipFree(&tmp);
}


__global__ void maxFilterSeparableMTHShMemKernel(CudaImage imgDst, const CudaImage imgSrc, CudaImage tmp)
{
  uint8_t max_val;
  int x = blockIdx.x*16 + threadIdx.x - 2;
  int y = blockIdx.y*16 + threadIdx.y - 2;

  __shared__ uint8_t MemShared[(N+4)*(N+4)];

  x = max(0,x);
  x = min(x, imgSrc.width-1);
  x = max(y,0);
  x = min(y, imgSrc.height-1);

  uint index = y*imgSrc.width + x;
  uint bindex =  threadIdx.y * blockDim.y + threadIdx.x;

  MemShared[bindex] = imgSrc.data[index];

  __syncthreads();

  // if ((threadIdx.x >= 2) && (threadIdx.x < (N+4) - 2) && (threadIdx.y >= 2) && (threadIdx.y < (N+4) - 2)){
  //   float sum = 0;
  //   for(int dy=-2; dy <= 2; dy++){
  //     for(int dx=-2; dx <= 2; dx++){
  //       float i = MemShared[bindex + (dy*blockDim.x) + dx];
  //       sum += i;
  //     }
  //   }
  //   imgDst.data[index]=sum/25;
  // }
  //__shared__ uint8_t MemShared_tmp[N+4][N+4];
  // if(i<2 || i>=imgSrc.width-2 || j<2 || j>=imgSrc.height-2)
  // {return;}
  //
  // uint shY = threadIdx.y + 2;
  // uint shX = threadIdx.x + 2;
  //
  // /*Only The boundary threads of Thread-Block will do extra effort of padding*/
  // if (threadIdx.x==0 || threadIdx.x==blocks.x-1 || threadIdx.y==0 || threadIdx.y==blocks.y-1){}
  // else{
  //   MemShared[shY][shX] = imgSrc.data[Idx];
  // }
  // __syncthreads();
  //
  max_val = MemShared[index];
  for (int32_t b = y-2; b < y+2; b++) {
    uint8_t value = max_val;
    if(b >= 0)
      value = MemShared[bindex + (b*blockDim.x) + x];
    if(value > max_val)
      max_val = value;
    }
  imgDst.data[index] = max_val;
  //
  // // __syncthreads();
  // // max = tmp.data[Idx];
  // // for (int32_t a = i-2; a < i+2; a++) {
  // //   uint8_t value = max;
  // //   if(a >= 0 && a < tmp.width)
  // //     value = tmp.data[j*tmp.pitch + a];
  // //   if(value > max)
  // //     max = value;
  // //   }
  // // imgDst.data[Idx] = max;



}

void maxFilterSeparableMTHShMem(lti::channel8 &res, const lti::channel8 &imgCpu, float &dt_ms)
{
    CudaImage imgSrc, imgDst, tmp;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    //uint N = 32;

    dim3 blocks;
    blocks.x = (imgCpu.columns()+N-1)/N;
    blocks.y = (imgCpu.rows()+N-1)/N;
    dim3 threads(N,N);

    imgSrc.width = imgDst.width =  tmp.width = imgCpu.columns();
    imgSrc.height = imgDst.height = tmp.height = imgCpu.rows();

    hipMallocPitch(&imgSrc.data, &imgSrc.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&imgDst.data, &imgDst.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&tmp.data, &tmp.pitch, imgCpu.columns(), imgCpu.rows());


    hipMemcpy2D(imgSrc.data, imgSrc.pitch, imgCpu.data(), imgCpu.columns(), imgCpu.columns(), imgCpu.rows(), hipMemcpyHostToDevice);

    hipEventRecord(event1, 0);
    maxFilterSeparableMTHShMemKernel<<<blocks,threads>>>(imgDst,imgSrc,tmp);
    hipEventRecord(event2, 0);


    hipEventSynchronize(event1);
    hipEventSynchronize(event2);

    hipEventElapsedTime(&dt_ms, event1, event2);

    hipMemcpy2D(res.data(), res.columns(), imgDst.data, imgDst.pitch, res.columns(), res.rows(), hipMemcpyDeviceToHost);

    hipFree(&imgSrc);
    hipFree(&imgDst);
    hipFree(&tmp);
}
