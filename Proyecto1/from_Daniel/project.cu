#include "hip/hip_runtime.h"
#include "project.hpp"
#include <cstdint>

struct CudaImage
{
    uint8_t *data;
    uint32_t width;
    uint32_t height;
    size_t pitch;
};

__global__ void maxFilterTrivialKernel(CudaImage imgDst, const CudaImage imgSrc)
{
    for(int32_t j=0; j<imgSrc.height; j++)
    {
        for(int32_t i=0; i<imgSrc.width; i++)
        {
            uint8_t max = imgSrc.data[j*imgSrc.pitch + i];

            for(int32_t a=i-2; a<i+2; a++)
            {
                for(int32_t b=j-2; b<j+2; b++)
                {
                    uint8_t value=max;
                    if(a >= 0 && a < imgSrc.width && b >= 0 && b < imgSrc.height)
                        value = imgSrc.data[b*imgSrc.pitch + a];
                    if(value > max)
                        max = value;
                }
            }
            imgDst.data[j*imgDst.pitch + i] = max;
        }
    }
}

void maxFilterTrivial(lti::channel8 &res, const lti::channel8 &imgCpu)
{
    CudaImage imgSrc, imgDst;

    imgSrc.width = imgDst.width = imgCpu.columns();
    imgSrc.height = imgDst.height = imgCpu.rows();

    hipMallocPitch(&imgSrc.data, &imgSrc.pitch, imgCpu.columns(), imgCpu.rows());
    hipMallocPitch(&imgDst.data, &imgDst.pitch, imgCpu.columns(), imgCpu.rows());

    hipMemcpy2D(imgSrc.data, imgSrc.pitch, imgCpu.data(), imgCpu.columns(), imgCpu.columns(), imgCpu.rows(), hipMemcpyHostToDevice);

    maxFilterTrivialKernel<<<1,1>>>(imgDst,imgSrc);

    hipMemcpy2D(res.data(), res.columns(), imgDst.data, imgDst.pitch, res.columns(), res.rows(), hipMemcpyDeviceToHost);
}
